#include <stdint.h>
#include <sys/time.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <assert.h>

extern "C" void compute2d_acc(uint8_t **, int, int, uint8_t *, uint32_t *);

#define BLOCK_SIZE 256

__global__ void run(uint8_t *input, uint32_t num_rows, uint32_t num_cols, uint32_t *thread_idxs, uint32_t *output) {
  uint64_t index = blockIdx.x * blockDim.x + threadIdx.x;
  uint32_t first_col = thread_idxs[index * 2];
  uint32_t second_col = thread_idxs[index * 2 + 1];
  uint32_t *our_output = output + index * 512;
  uint8_t *input_ptr = input;

  uint32_t counts[512] = {0};

  for (uint32_t i = 0; i < num_rows; i++) {
    uint8_t metric = *input_ptr;
    input_ptr += 1;
    uint8_t counts_idx = (input_ptr[first_col] << 4) | input_ptr[second_col];
    counts[2 * counts_idx] += metric;
    counts[2 * counts_idx + 1]++;
    input_ptr += num_cols;
  }
  for (uint32_t i = 0; i < 512; i++) {
    our_output[i] = counts[i];
  }
}

void compute2d_acc(uint8_t **cols, int num_rows, int num_cols, uint8_t *metric, uint32_t *stats) {
  assert(hipSetDevice(0) == hipSuccess);

  uint64_t input_size = (sizeof(uint8_t) + sizeof(uint8_t) * num_cols) * ((uint64_t)num_rows);
  uint8_t *input = (uint8_t *)malloc(input_size);
  // column to row
  #pragma omp parallel for
  for (int i = 0; i < num_rows; i++) {
    uint8_t *cur = input + (num_cols + 1) * i;
    *cur++ = metric[i];
    for (int j = 0; j < num_cols; j++) {
      *cur++ = cols[j][i];
    }
  }

  int num_pairs = num_cols * (num_cols + 1) / 2;
  int num_blocks = (num_pairs + BLOCK_SIZE - 1) / BLOCK_SIZE;
  int num_threads = num_blocks * BLOCK_SIZE;
  int thread_idxs_size = sizeof(uint32_t) * 2 * num_threads;
  uint32_t *thread_idxs = (uint32_t *)calloc(thread_idxs_size, 1);
  int pair_cnt = 0;
  for (int i = 0; i < num_cols; i++) {
    for (int j = i; j < num_cols; j++) {
      thread_idxs[2 * pair_cnt] = i;
      thread_idxs[2 * pair_cnt + 1] = j;
      pair_cnt++;
    }
  }

  uint8_t *input_dev;
  uint32_t *thread_idxs_dev;
  uint32_t *output_dev;
  int output_size = sizeof(uint32_t) * 512 * num_threads;
  assert(hipMalloc((void **) &output_dev, output_size) == hipSuccess);
  assert(hipMalloc((void **) &input_dev, input_size) == hipSuccess);
  assert(hipMalloc((void **) &thread_idxs_dev, thread_idxs_size) == hipSuccess);
  hipMemcpy(input_dev, input, input_size, hipMemcpyHostToDevice);
  hipMemcpy(thread_idxs_dev, thread_idxs, thread_idxs_size, hipMemcpyHostToDevice);

  run<<<num_blocks, BLOCK_SIZE>>>(input_dev, num_rows, num_cols, thread_idxs_dev, output_dev);
  assert(hipDeviceSynchronize() == hipSuccess);

  hipMemcpy(stats, output_dev, 512 * sizeof(uint32_t) * num_pairs, hipMemcpyDeviceToHost);

  hipFree(output_dev);
  hipFree(input_dev);
  hipFree(thread_idxs_dev);
  free(input);
  free(thread_idxs);
}
