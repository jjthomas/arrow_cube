#include <stdint.h>
#include <sys/time.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>
#include <assert.h>

extern "C" void compute2d_acc(uint8_t **, int, int, uint8_t *, uint32_t *);

#define BLOCK_SIZE 256

__global__ void run(uint8_t *input, uint32_t num_rows, uint32_t num_cols, uint32_t *block_idxs, uint32_t *output) {
  uint64_t index = blockIdx.x;
  uint32_t first_col = block_idxs[index * 2];
  uint32_t second_col = block_idxs[index * 2 + 1];
  uint32_t *our_output = output + index * 512;
  uint8_t *metric = input;
  uint8_t *col1 = input + (first_col + 1) * num_rows;
  uint8_t *col2 = input + (second_col + 1) * num_rows;

  __shared__ uint32_t counts[512];
  for (int i = threadIdx.x; i < 512; i += blockDim.x) {
    counts[i] = 0;
  }
  __syncthreads();

  for (uint32_t i = threadIdx.x; i < num_rows; i += blockDim.x) {
    uint8_t counts_idx = (col1[i] << 4) | col2[i];
    atomicAdd(&counts[2 * counts_idx], metric[i]);
    atomicAdd(&counts[2 * counts_idx + 1], 1);
  }
  __syncthreads();

  for (int i = threadIdx.x; i < 512; i += blockDim.x) {
    our_output[i] = counts[i];
  }
}

void compute2d_acc(uint8_t **cols, int num_rows, int num_cols, uint8_t *metric, uint32_t *stats) {
  assert(hipSetDevice(0) == hipSuccess);

  uint64_t input_size = ((uint64_t)num_rows) * (num_cols + 1);
  uint8_t *input = (uint8_t *)malloc(input_size);
  memcpy(input, metric, num_rows);
  #pragma omp parallel for
  for (int i = 0; i < num_cols; i++) {
    memcpy(input + (i + 1) * num_rows, cols[i], num_rows);
  }

  int num_pairs = num_cols * (num_cols + 1) / 2;
  int block_idxs_size = sizeof(uint32_t) * 2 * num_pairs;
  uint32_t *block_idxs = (uint32_t *)calloc(block_idxs_size, 1);
  int pair_cnt = 0;
  for (int i = 0; i < num_cols; i++) {
    for (int j = i; j < num_cols; j++) {
      block_idxs[2 * pair_cnt] = i;
      block_idxs[2 * pair_cnt + 1] = j;
      pair_cnt++;
    }
  }

  uint8_t *input_dev;
  uint32_t *block_idxs_dev;
  uint32_t *output_dev;
  int output_size = sizeof(uint32_t) * 512 * num_pairs;
  assert(hipMalloc((void **) &output_dev, output_size) == hipSuccess);
  assert(hipMalloc((void **) &input_dev, input_size) == hipSuccess);
  assert(hipMalloc((void **) &block_idxs_dev, block_idxs_size) == hipSuccess);
  hipMemcpy(input_dev, input, input_size, hipMemcpyHostToDevice);
  hipMemcpy(block_idxs_dev, block_idxs, block_idxs_size, hipMemcpyHostToDevice);

  run<<<num_pairs, BLOCK_SIZE>>>(input_dev, num_rows, num_cols, block_idxs_dev, output_dev);
  assert(hipDeviceSynchronize() == hipSuccess);

  hipMemcpy(stats, output_dev, output_size, hipMemcpyDeviceToHost);

  hipFree(output_dev);
  hipFree(input_dev);
  hipFree(block_idxs_dev);
  free(input);
  free(block_idxs);
}
